
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void oddEvenSort(int *arr, int n)
{
    int temp;
    int i, idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (i = 0; i < n; i++)
    {
        if (idx % 2 == 0) // Even phase
        {
            if (i % 2 == 0 && idx < n-1)
            {
                if (arr[idx] > arr[idx+1])
                {
                    temp = arr[idx];
                    arr[idx] = arr[idx+1];
                    arr[idx+1] = temp;
                }
            }
        }
        else // Odd phase
        {
            if (i % 2 == 1 && idx < n-1)
            {
                if (arr[idx] > arr[idx+1])
                {
                    temp = arr[idx];
                    arr[idx] = arr[idx+1];
                    arr[idx+1] = temp;
                }
            }
        }
    }
}

int main()
{
    int n;
    printf("Enter the number of elements to sort: ");
    scanf("%d", &n);

    int *arr, *d_arr;
    arr = (int *) malloc(n * sizeof(int));

    printf("Enter %d elements to sort: ", n);
    for (int i = 0; i < n; i++)
    {
        scanf("%d", &arr[i]);
    }

    hipMalloc(&d_arr, n * sizeof(int));
    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    oddEvenSort<<<blocksPerGrid, threadsPerBlock>>>(d_arr, n);

    hipMemcpy(arr, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);

    printf("Sorted array: ");
    for (int i = 0; i < n; i++)
    {
        printf("%d ", arr[i]);
    }

    hipFree(d_arr);
    free(arr);

    return 0;
}
