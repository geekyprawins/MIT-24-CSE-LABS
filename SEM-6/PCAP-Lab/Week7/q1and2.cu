#include<stdio.h>
#include "hip/hip_runtime.h"


__global__ void vectorAdd_q1A(float *A, float *B,float *C)
{
	int tid=blockIdx.x;
		C[tid]=A[tid]+B[tid];
}

__global__ void vectorAdd_q1B(float *A, float *B,float *C)
{
	int tid=threadIdx.x;
		C[tid]=A[tid]+B[tid];
}

__global__ void vectorAdd_q2(float *A, float *B,float *C,int n)
{
	int tid=blockIdx.x*blockDim.x+threadIdx.x;

	if(tid<n)
		C[tid]=A[tid]+B[tid];
}

int main(void) 
{
	int n;

	printf("Enter number of elements:\t");
	scanf("%d",&n);

	float *h_A,*h_B,*h_C;
	float *d_A, *d_B, *d_C;
	
	int size = sizeof(float)*n;

	h_A=(float*)malloc(sizeof(size));
	h_B=(float*)malloc(sizeof(size));
	h_C=(float*)malloc(sizeof(size));

	hipMalloc((void **)&d_A, size);
	hipMalloc((void **)&d_B, size);
	hipMalloc((void **)&d_C, size);

	printf("Enter vector 1:\n");
	for(int i=0;i<n;i++)
		scanf("%f",&h_A[i]);

	printf("Enter vector 2:\n");
	for(int i=0;i<n;i++)
		scanf("%f",&h_B[i]);


	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	
	dim3 dimGrid_q1A(n,1,1);
	dim3 dimBlock_q1A(1,1,1);

	vectorAdd_q1A<<<dimGrid_q1A,dimBlock_q1A>>>(d_A,d_B,d_C);
	
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	printf("Resultant Vector Q1_A:\n");
	for(int i=0;i<n;i++)
		printf("%f\t",h_C[i]);

	printf("\n");

	dim3 dimGrid_q1B(1,1,1);
	dim3 dimBlock_q1B(n,1,1);

	vectorAdd_q1B<<<dimGrid_q1B,dimBlock_q1B>>>(d_A,d_B,d_C);
	
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	printf("Resultant Vector Q1_B:\n");
	for(int i=0;i<n;i++)
		printf("%f\t",h_C[i]);

	printf("\n");

	dim3 dimGrid_q2(ceil(n/256.0),1,1);
	dim3 dimBlock_q2(256,1,1);

	vectorAdd_q2<<<dimGrid_q2,dimBlock_q2>>>(d_A,d_B,d_C,n);
	
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	printf("Resultant Vector Q2:\n");
	for(int i=0;i<n;i++)
		printf("%f\t",h_C[i]);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	return 0;
}
