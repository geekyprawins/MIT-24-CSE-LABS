#include<stdio.h>
#include "hip/hip_runtime.h"



__global__ void vectorSin(float *A, float *B,int n)
{
	int tid=blockIdx.x*blockDim.x+threadIdx.x;

	if(tid<n)
		B[tid]=sin(A[tid]);
}

int main(void) 
{
	int n;

	printf("Enter number of elements:\t");
	scanf("%d",&n);

	float *h_A,*h_B;
	float *d_A, *d_B;
	
	int size = sizeof(float)*n;

	h_A=(float*)malloc(sizeof(size));
	h_B=(float*)malloc(sizeof(size));

	hipMalloc((void **)&d_A, size);
	hipMalloc((void **)&d_B, size);

	printf("Enter input vector:\n");
	for(int i=0;i<n;i++)
		scanf("%f",&h_A[i]);

	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	
	dim3 dimGrid(ceil(n/256.0),1,1);
	dim3 dimBlock(256,1,1);

	vectorSin<<<dimGrid,dimBlock>>>(d_A,d_B,n);
	
	hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);

	printf("Resultant Sin vector:\n");
	for(int i=0;i<n;i++)
		printf("%f\t",h_B[i]);

	hipFree(d_A);
	hipFree(d_B);
	return 0;
}
