#include<stdio.h>
#include "hip/hip_runtime.h"



__global__ void Convolution_1D(float *N, float *M,float *P,int width,int mask_width)
{
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	float pValue=0;
	int startPoint= tid - (mask_width/2);

	for(int j=0;j<mask_width;j++)
	{
		if((startPoint+j>=0)&&(startPoint+j<width))
			pValue+=N[startPoint+j]*M[j];
	}
	P[tid]=pValue;
}

int main(void) 
{
	int width,mask_width;

	printf("Enter width:\t");
	scanf("%d",&width);

	printf("Enter mask_width:\t");
	scanf("%d",&mask_width);

	float *h_N,*h_P,*h_M;
	float *d_N,*d_P,*d_M;

	h_N=(float*)malloc(sizeof(float)*width);
	h_P=(float*)malloc(sizeof(float)*width);
	h_M=(float*)malloc(sizeof(float)*mask_width);

	hipMalloc((void **)&d_N,sizeof(float)*width);
	hipMalloc((void **)&d_P,sizeof(float)*width);
	hipMalloc((void **)&d_M,sizeof(float)*mask_width);

	printf("Enter N:\n");
	for(int i=0;i<width;i++)
		scanf("%f",&h_N[i]);

	printf("Enter M:\n");
	for(int i=0;i<mask_width;i++)
		scanf("%f",&h_M[i]);


	hipMemcpy(d_N, h_N,sizeof(float)*width, hipMemcpyHostToDevice);
	hipMemcpy(d_M, h_M,sizeof(float)*mask_width, hipMemcpyHostToDevice);
	
	dim3 dimGrid(ceil(width/256.0),1,1);
	dim3 dimBlock(256,1,1);

	Convolution_1D<<<dimGrid,dimBlock>>>(d_N,d_M,d_P,width,mask_width);
	
	hipMemcpy(h_P, d_P,sizeof(float)*width, hipMemcpyDeviceToHost);

	printf("Resultant P:\n");
	for(int i=0;i<width;i++)
		printf("%f\t",h_P[i]);

	hipFree(d_N);
	hipFree(d_P);
	hipFree(d_M);
	return 0;
}
