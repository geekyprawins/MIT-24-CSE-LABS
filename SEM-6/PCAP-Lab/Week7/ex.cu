#include<stdio.h>
#include "hip/hip_runtime.h"


__global__ void add(int *a, int *b, int *c)
{
	*c = *a + *b;
}

int main(void) 
{
	int a, b, c;
	int *d_a, *d_b, *d_c;
	
	int size = sizeof(int);

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	a = 3;
	b = 5;

	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
	
	add<<<1,1>>>(d_a, d_b, d_c);
	
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
	printf("Result : %d",c);
	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}