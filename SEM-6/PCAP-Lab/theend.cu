#include<stdio.h>
#include "hip/hip_runtime.h"

__global__ void calc(char* a, char* res, int* s, int n, char dir){
  int bi = blockIdx.x, j = threadIdx.x, i = threadIdx.y, tid = i * 2 + j, first = s[bi], last = s[bi + 1], size = last - first - 1, m = n % size;
  res[first + size] = ' ';
  if(dir == 'R' && tid < size) res[first + tid] = a[first + (tid - m + size) % size];
  if(dir == 'L' && tid < size) res[first + tid] = a[first + (tid + m) % size];
}
int main(){
  char a[100], *d_a, res[100], *d_res, dir;
  int n;
  printf("Enter the sentence: ");
  fgets(a, sizeof(a), stdin); // need to give an extra space along with the last word in the sentence.
  printf("Enter the direction: ");
  scanf("%c", &dir);
  printf("Enter the value of n: ");
  scanf("%d", &n);
  int st[100], len = 0, i = 0, *d_st, maxi = 0;
  st[0] = 0;
  for(; a[len] != '\n'; len++){
    if(a[len] == ' '){
      st[++i] = len + 1;
      printf(": %d\n", st[i] - st[i - 1] - 1);
      if(maxi < st[i] - st[i - 1] - 1) maxi = st[i] - st[i - 1] - 1;
    }
    else printf("%c", a[len]);
  }
  printf("Largest word length is: %d\n", maxi);
  hipMalloc((void**)&d_a, sizeof(char) * len);
  hipMalloc((void**)&d_res, sizeof(char) * len);
  hipMalloc((void**)&d_st, sizeof(int) * (i + 1));
  hipMemcpy(d_a, &a, sizeof(char) * len, hipMemcpyHostToDevice);
  hipMemcpy(d_st, &st, sizeof(int) * (i + 1), hipMemcpyHostToDevice);
  dim3 bl(2, maxi / 2 + (maxi % 2 > 0));
  calc<<<i, bl >>> (d_a, d_res, d_st, n, dir);
  hipMemcpy(&res, d_res, sizeof(char) * len, hipMemcpyDeviceToHost);
  res[len] = '\0';
  printf("Resultant string is %s\n", res);
  hipFree(d_st);
  hipFree(d_a);
  hipFree(d_res);
  return 0;
}