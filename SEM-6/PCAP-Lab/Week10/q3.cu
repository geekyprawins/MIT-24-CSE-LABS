
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void complement_kernel(int* A, int* B, int M, int N)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int num, binary_num = 0, base = 1, mask;
    if (i < M && j < N) {
        if (i == 0 || i == M-1 || j == 0 || j == N-1) {
            B[i*N+j] = A[i*N+j];
        } else {
            num = A[i*N+j];
            mask = ~0;
            while (num & mask) {
                mask <<= 1;
            }
            num = ~num & ~mask;
            while (num > 0) {
                binary_num += (num % 2) * base;
                num /= 2;
                base *= 10;
            }
            B[i*N+j] = binary_num;
        }
    }
}

int main()
{
    int M, N;
    printf("Enter the number of rows of the matrix: ");
    scanf("%d", &M);
    printf("Enter the number of columns of the matrix: ");
    scanf("%d", &N);

    int A[M*N];
    printf("Enter the elements of the matrix:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            scanf("%d", &A[i*N+j]);
        }
    }

    int B[M*N];

    // Allocate memory on the GPU
    int *d_A, *d_B;
    hipMalloc((void **)&d_A, M * N * sizeof(int));
    hipMalloc((void **)&d_B, M * N * sizeof(int));

    // Copy input data to the GPU
    hipMemcpy(d_A, A, M * N * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 block_size(16, 16);
    dim3 num_blocks((N + block_size.x - 1) / block_size.x, (M + block_size.y - 1) / block_size.y);
    complement_kernel<<<num_blocks, block_size>>>(d_A, d_B, M, N);

    // Copy output data from the GPU
    hipMemcpy(B, d_B, M * N * sizeof(int), hipMemcpyDeviceToHost);

    // Print the output matrix
    printf("Output matrix:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", B[i*N+j]);
        }
        printf("\n");
    }

    // Free memory on the GPU
    hipFree(d_A);
    hipFree(d_B);

    return 0;
}
