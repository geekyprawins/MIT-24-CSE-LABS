#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void replace_rows(float *A, int num_rows, int num_cols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= num_rows) return;

    for (int j = 0; j < num_cols; j++) {
        int idx = row * num_cols + j;
        int power = row + 1;
        A[idx] = powf(A[idx], power);
    }
}

int main() {
    int M, N;
    printf("Enter no of rows and columns : ");
    scanf("%d %d", &M, &N);

    float *A = (float*)malloc(sizeof(float) * M * N);
    printf("Enter the matrix :\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            scanf("%f", &A[i * N + j]);
        }
    }

    float *d_A;
    hipMalloc(&d_A, sizeof(float) * M * N);
    hipMemcpy(d_A, A, sizeof(float) * M * N, hipMemcpyHostToDevice);

    replace_rows<<<(M + 255) / 256, 256>>>(d_A, M, N);

    hipMemcpy(A, d_A, sizeof(float) * M * N, hipMemcpyDeviceToHost);

    printf("Modified matrix : \n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            printf("%g \t", A[i * N + j]);
        }
        printf("\n");
    }

    free(A);
    hipFree(d_A);
    return 0;
}
