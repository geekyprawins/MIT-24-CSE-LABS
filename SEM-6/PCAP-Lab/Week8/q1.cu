#include <stdio.h>
#include <hip/hip_runtime.h>

#define MAX_SENTENCE_LENGTH 100
#define MAX_WORD_LENGTH 10
#define BLOCK_SIZE 128

__global__ void countWordKernel(char* sentence_h, int* wordCount_h, char* word_h) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    while (index < MAX_SENTENCE_LENGTH) {
        int i = 0;
        while (i < MAX_WORD_LENGTH && word_h[i] != '\0' && sentence_h[index + i] == word_h[i]) {
            i++;
        }
        if (i == MAX_WORD_LENGTH || word_h[i] == '\0') {
            atomicAdd(wordCount_h, 1);
        }
        index += stride;
    }
}

int main() {

    char sentence_h[MAX_SENTENCE_LENGTH];
    char word_h[MAX_WORD_LENGTH];
    int wordCount_h;

    printf("Enter a sentence: ");
    scanf("%[^\n]s",sentence_h);
    printf("Enter a word: ");
    scanf("%s", word_h);

    char* sentence_d;
    char* word_d;
    int* wordCount_d;

    hipMalloc(&sentence_d, MAX_SENTENCE_LENGTH * sizeof(char));
    hipMalloc(&word_d, MAX_WORD_LENGTH * sizeof(char));
    hipMalloc(&wordCount_d, sizeof(int));
    
    hipMemcpy(sentence_d, sentence_h, MAX_SENTENCE_LENGTH * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(word_d, word_h, MAX_WORD_LENGTH * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(wordCount_d, &wordCount_h, sizeof(int), hipMemcpyHostToDevice);
    
    countWordKernel<<<(MAX_SENTENCE_LENGTH + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(sentence_d, wordCount_d, word_d);

    hipMemcpy(&wordCount_h, wordCount_d, sizeof(int), hipMemcpyDeviceToHost);

    printf("The word '%s' appears %d times in the sentence.\n", word_h, wordCount_h);

    hipFree(sentence_d);
    hipFree(word_d);
    hipFree(wordCount_d);

    return 0;

}
