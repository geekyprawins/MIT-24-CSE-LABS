#include <stdio.h>
#include "hip/hip_runtime.h"



__global__ void pattern(char *str, char *ans, int len)
{
	int tid=blockIdx.x;
	int idx = (len*(len+1)/2) - ((len-tid)*(len+1-tid)/2);
	for(int i=0; i<(len-tid); i++)
		ans[idx+i] = str[i];
}

int main(void) {
    char str_h[10], *str_d;
	char *ans_h, *ans_d;
    printf("Enter the string: ");
    scanf("%s", str_h);
    int len = strlen(str_h);
	ans_h = (char*)malloc(((len*(len+1))/2)*sizeof(char));	
	hipMalloc((void **)&str_d, len*sizeof(char));
	hipMalloc((void **)&ans_d, ((len*(len+1))/2)*sizeof(char));
	hipMemcpy(str_d, str_h, len*sizeof(char), hipMemcpyHostToDevice);
	pattern<<<len,1>>>(str_d, ans_d, len);
	hipMemcpy(ans_h, ans_d, ((len*(len+1))/2)*sizeof(char), hipMemcpyDeviceToHost);
	printf("Final string is: %s.\n", ans_h);
	hipFree(str_d);
	hipFree(ans_d);
	return 0;
}