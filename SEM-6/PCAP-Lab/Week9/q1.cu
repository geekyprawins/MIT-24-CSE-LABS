#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void add_matrices_row(int *a, int *b, int *c, int n, int m)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        for (int j = 0; j < m; j++)
        {
            c[i * m + j] = a[i * m + j] + b[i * m + j];
        }
    }
}

__global__ void add_matrices_col(int *a, int *b, int *c, int n, int m)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j < m)
    {
        for (int i = 0; i < n; i++)
        {
            c[i * m + j] = a[i * m + j] + b[i * m + j];
        }
    }
}

__global__ void add_matrices_elem(int *a, int *b, int *c, int n, int m)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n * m)
    {
        int i = idx / m;
        int j = idx % m;
        c[i * m + j] = a[i * m + j] + b[i * m + j];
    }
}

int main()
{
    int n, m;
    printf("Enter the dimensions of the matrices (n m): ");
    scanf("%d %d", &n, &m);
    int size = sizeof(int) * n * m;

    int *a = (int *)malloc(size);
    int *b = (int *)malloc(size);
    int *c = (int *)malloc(size);
    int *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    printf("Enter the first matrix:\n");
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < m; j++)
        {
            scanf("%d", &a[i * m + j]);
        }
    }

    printf("Enter the second matrix:\n");
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < m; j++)
        {
            scanf("%d", &b[i * m + j]);
        }
    }

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    int block_size = 256;
    int grid_size;

    int option;
    printf("Enter option(1. Row-wise 2. Column-wise 3. Element-wise) :\n");
    scanf("%d", &option);
    switch (option)
    {
    case 1:
        grid_size = (n + block_size - 1) / block_size;
        // Compute each row of the resultant matrix using one thread
        add_matrices_row<<<grid_size, block_size>>>(d_a, d_b, d_c, n, m);

        hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
        printf("Result matrix (row-wise addition):\n");
        for (int i = 0; i < n; i++)
        {
            for (int j = 0; j < m; j++)
            {
                printf("%d ", c[i * m + j]);
            }
            printf("\n");
        }
        break;
    case 2:
        // Compute each column of the resultant matrix using one thread
        grid_size = (m + block_size - 1) / block_size;
        add_matrices_col<<<grid_size, block_size>>>(d_a, d_b, d_c, n, m);

        hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
        printf("Result matrix (column-wise addition):\n");
        for (int i = 0; i < n; i++)
        {
            for (int j = 0; j < m; j++)
            {
                printf("%d ", c[i * m + j]);
            }
            printf("\n");
        }
        break;
    case 3:
        // Compute each element of the resultant matrix using one thread
        grid_size = (n * m + block_size - 1) / block_size;
        add_matrices_elem<<<grid_size, block_size>>>(d_a, d_b, d_c, n, m);

        hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
        printf("Result matrix (element-wise addition):\n");
        for (int i = 0; i < n; i++)
        {
            for (int j = 0; j < m; j++)
            {
                printf("%d ", c[i * m + j]);
            }
            printf("\n");
        }
        break;
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);
    return 0;
}