#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void multiply_matrices_row(int *a, int *b, int *c, int n, int m, int k)
{
        int row = blockIdx.x;
        for (int col = 0; col < k; col++)
        {
                int sum = 0;
                for (int i = 0; i < m; i++)
                {
                        sum += a[row * m + i] * b[i * k + col];
                }
                c[row * k + col] = sum;
        }
}

__global__ void multiply_matrices_col(int *a, int *b, int *c, int n, int m, int k)
{
        int col = blockIdx.x;
        for (int row = 0; row < n; row++)
        {
                int sum = 0;
                for (int i = 0; i < m; i++)
                {
                        sum += a[row * m + i] * b[i * k + col];
                }
                c[row * k + col] = sum;
        }
}

__global__ void multiply_matrices_elem(int *a, int *b, int *c, int n, int m, int k)
{
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        if (tid < n * k)
        {
                int row = tid / k;
                int col = tid % k;
                int sum = 0;
                for (int i = 0; i < m; i++)
                {
                        sum += a[row * m + i] * b[i * k + col];
                }
                c[row * k + col] = sum;
        }
}

int main()
{
        int *a, *b, *c;
        int *d_a, *d_b, *d_c;
        int n, m, k;

        // Read input matrices dimensions
        printf("Enter dimensions of matrix A (n x m): ");
        scanf("%d%d", &n, &m);
        printf("Enter dimensions of matrix B (m x k): ");
        scanf("%d%d", &m, &k);

        // Allocate memory on host and device
        int size_a = n * m * sizeof(int);
        int size_b = m * k * sizeof(int);
        int size_c = n * k * sizeof(int);
        a = (int *)malloc(size_a);
        b = (int *)malloc(size_b);
        c = (int *)malloc(size_c);
        hipMalloc(&d_a, size_a);
        hipMalloc(&d_b, size_b);
        hipMalloc(&d_c, size_c);

        // Read input matrix data and copy to device
        printf("Enter matrix A: \n");
        for (int i = 0; i < n; i++)
        {
                for (int j = 0; j < m; j++)
                {
                        scanf("%d", &a[i * m + j]);
                }
        }
        printf("Enter matrix B: \n");
        for (int i = 0; i < m; i++)
        {
                for (int j = 0; j < k; j++)
                {
                        scanf("%d", &b[i * k + j]);
                }
        }
        hipMemcpy(d_a, a, size_a, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, size_b, hipMemcpyHostToDevice);

        dim3 grid_size_row(n, 1, 1);
        dim3 block_size_row(1, 1, 1);
        dim3 grid_size_col(k, 1, 1);
        dim3 block_size_col(1, 1, 1);
        dim3 grid_size_elem((n * k + 255) / 256, 1, 1);
        dim3 block_size_elem(256, 1, 1);

        int option;
        printf("Enter option(1. Row-wise 2. Column-wise 3. Element-wise) :\n");
        scanf("%d", &option);
        switch (option)
        {
        case 1:
                // Compute matrix multiplication row-wise using one thread per row

                multiply_matrices_row<<<grid_size_row, block_size_row>>>(d_a, d_b, d_c, n, m, k);
                hipMemcpy(c, d_c, size_c, hipMemcpyDeviceToHost);
                printf("Matrix multiplication row-wise: \n");
                for (int i = 0; i < n; i++)
                {
                        for (int j = 0; j < k; j++)
                        {
                                printf("%d ", c[i * k + j]);
                        }
                        printf("\n");
                }
                printf("\n");
                break;
        case 2:
                // Compute matrix multiplication column-wise using one thread per column

                multiply_matrices_col<<<grid_size_col, block_size_col>>>(d_a, d_b, d_c, n, m, k);
                hipMemcpy(c, d_c, size_c, hipMemcpyDeviceToHost);
                printf("Matrix multiplication column-wise: \n");
                for (int i = 0; i < n; i++)
                {
                        for (int j = 0; j < k; j++)
                        {
                                printf("%d ", c[i * k + j]);
                        }
                        printf("\n");
                }
                printf("\n");

                break;
        case 3:
                // Compute matrix multiplication element-wise using one thread per element

                multiply_matrices_elem<<<grid_size_elem, block_size_elem>>>(d_a, d_b, d_c, n, m, k);
                hipMemcpy(c, d_c, size_c, hipMemcpyDeviceToHost);
                printf("Matrix multiplication element-wise: \n");
                for (int i = 0; i < n; i++)
                {
                        for (int j = 0; j < k; j++)
                        {
                                printf("%d ", c[i * k + j]);
                        }
                        printf("\n");
                }
                printf("\n");

                break;
        }

        // Free memory on host and device
        free(a);
        free(b);
        free(c);
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);

        return 0;
}