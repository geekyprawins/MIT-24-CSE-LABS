#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include<>
#include<stdlib.h>

__global__ void transpose(int *a, int *t){
    int n=threadIdx.x, m=blockIdx.x, size=blockDim.x, size1=gridDim.x;
    t[n*size+m]=a[m*size+n];
}

int main(void){
    int *a, *t, m, n, i, j;
    int *d_a, *d_t;

    printf("Enter m:");
    scanf("%d", &m);
    printf("Enter n:");
    scanf("%d", &n);

    int size=sizeof(int)*m*n;

    a=(int*)malloc(m*n*sizeof(int));
    t=(int*)malloc(m*n*sizeof(int));

    printf("Enter input matrix: ");

    for(i=0; i<m*n; i++){
        scanf("%d", &a[i]);
    }

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_t, size);
    
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    transpose<<<m,n>>>(d_a,d_t);
    hipMemcpy(t,d_t, size, hipMemcpyDeviceToHost);
    printf("Result vector is: \n");

    for(int i=0; i<n; i++){
        for(int j=0; j<m; j++){
            printf("%d\t", t[i*m+j]);
        }
        printf("\n");
    }

    getChar();
    hipFree(d_a);
    hipFree(d_t);
    
    return 0;
}