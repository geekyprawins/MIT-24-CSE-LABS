// Matrix multiplication of 4x4 matrix #include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#define BLOCK_WIDTH 2
#define TILE_WIDTH 2
#define WIDTH 4

__global__ void MatMulElementThreadShared(int *a, int *b, int *c)
{
    __shared__ int MDs[TILE_WIDTH][TILE_WIDTH];
    __shared__ int NDs[TILE_WIDTH][TILE_WIDTH];
    int m;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;
    int Pvalue = 0;
    for (m = 0; m < WIDTH / TILE_WIDTH; m++)
    {
        MDs[ty][tx] = a[Row * WIDTH + m * TILE_WIDTH + tx];
        NDs[ty][tx] = b[(m * TILE_WIDTH + ty) * WIDTH + Col];
        __syncthreads();
        for (int k = 0; k < TILE_WIDTH; k++)
        {
            Pvalue += MDs[ty][k] * NDs[k][tx];
        }
        __syncthreads();
    }
    c[Row * WIDTH + Col] = Pvalue;
}

int main()
{
    int *matA, *matB, *matProd;
    int *da, *db, *dc;
    printf("\n== Enter elements of Matrix A (4x4)==\n");
    matA = (int *)malloc(sizeof(int) * WIDTH * WIDTH);
    for (int i = 0; i < WIDTH * WIDTH; i++)
    {
        scanf("%d", &matA[i]);
    }
    printf("\n== Enter elements of Matrix B (4x4)=\n");
    matB = (int *)malloc(sizeof(int) * WIDTH * WIDTH);
    for (int i = 0; i < WIDTH * WIDTH; i++)
    { 
        scanf("%d", &matB[i]);
    }
   
    matProd = (int *)malloc(sizeof(int) * WIDTH * WIDTH);
    hipMalloc((void **)&da, sizeof(int) * WIDTH * WIDTH);
    hipMalloc((void **)&db, sizeof(int) * WIDTH * WIDTH);
    hipMalloc((void **)&dc, sizeof(int) * WIDTH * WIDTH);
    hipMemcpy(da, matA, sizeof(int) * WIDTH * WIDTH, hipMemcpyHostToDevice);
    hipMemcpy(db, matB, sizeof(int) * WIDTH * WIDTH, hipMemcpyHostToDevice);
    int NumBlocks = WIDTH / BLOCK_WIDTH;
    dim3 grid_conf(NumBlocks, NumBlocks);
    dim3 block_conf(BLOCK_WIDTH, BLOCK_WIDTH);
    MatMulElementThreadShared<<<grid_conf, block_conf>>>(da, db, dc);
    hipMemcpy(matProd, dc, sizeof(int) * WIDTH * WIDTH, hipMemcpyDeviceToHost);
    printf("\n--Result of Multiplication=-\n");
    printf("----------------\n");
    for (int i = 0; i < 4; i++)
    {
        for (int j = 0; j < 4; j++)
        {
            printf("%6d", matProd[i * 4 + j]);
        }
        printf("\n");
    }

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    free(matA);
    free(matB);
    free(matProd);

    return 0;
}
