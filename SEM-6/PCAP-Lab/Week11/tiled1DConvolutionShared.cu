
#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREAD_BLOCK_SIZE 256
#define MASK_WIDTH 3

__global__ void convolution_kernel(float* N, float* M, float* P, int width, int mask_width) {
    // Determine the index of the current thread
    int thread_idx = threadIdx.x;
    int block_idx = blockIdx.x;
    int block_dim = blockDim.x;

    // Compute the start and end index of the current tile
    int tile_start = block_idx * block_dim - mask_width/2;
    // int tile_end = tile_start + block_dim - 1 + mask_width;

    // Allocate shared memory for the tile
    __shared__ float tile_data[THREAD_BLOCK_SIZE + MASK_WIDTH - 1];

    // Load data from global memory into shared memory
    for (int i = thread_idx - mask_width/2; i < block_dim + mask_width/2; i += block_dim) {
        int index = tile_start + i;
        if (index >= 0 && index < width) {
            tile_data[i + mask_width/2] = N[index];
        } else {
            tile_data[i + mask_width/2] = 0;
        }
    }

    // Synchronize to ensure all threads have finished loading data into shared memory
    __syncthreads();

    // Compute the output of the current thread
    float output = 0;
    for (int i = 0; i < mask_width; i++) {
        output += tile_data[thread_idx + i] * M[i];
    }

    // Store the output in global memory
    if (thread_idx < block_dim && tile_start + thread_idx >= 0 && tile_start + thread_idx < width) {
        P[tile_start + thread_idx] = output;
    }
}



int main() {
    // Define the input and mask arrays
 

    float N[1000],  P[1000];
    int n;
    // Initialize the input and mask arrays
    printf("Enter width of main array: ");
    scanf("%d",&n);
    printf("enter elements of the main array: ");
    for(int i = 0;i<n;i++){
        scanf("%f",&N[i]);
    }
       int mask_width;
    printf("Enter width of mask array: \n");
    scanf("%d",&mask_width);
   float M[mask_width];
        printf("enter elements of the mask array: ");
    for(int i = 0;i<mask_width;i++){
        scanf("%f",&M[i]);
    }
    // Allocate device memory for the input, mask, and output arrays
    float *d_N, *d_M, *d_P;
    hipMalloc(&d_N, n * sizeof(float));
    hipMalloc(&d_M, mask_width * sizeof(float));
    hipMalloc(&d_P, n * sizeof(float));

    // Copy the input and mask arrays from host to device
    hipMemcpy(d_N, N, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_M, M, mask_width * sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel with the appropriate arguments
    int num_blocks = (n + THREAD_BLOCK_SIZE - 1) / THREAD_BLOCK_SIZE;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsedTime;
    
    
    hipEventRecord(start,0);
    
    convolution_kernel<<<num_blocks, THREAD_BLOCK_SIZE>>>(d_N, d_M, d_P, n, mask_width);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    elapsedTime = 0;
    hipEventElapsedTime(&elapsedTime, start, stop);
    // Copy the output array from device to host
    hipMemcpy(P, d_P, n * sizeof(float), hipMemcpyDeviceToHost);

    // Print the output array
    for (int i = 0; i < n; i++) {
        printf("%f ", P[i]);
    }
    printf("\n");
printf("\nTotal Time Taken(Shared Memory): %f\n", elapsedTime);
    // Free device memory
    hipFree(d_N);
    hipFree(d_M);
    hipFree(d_P);

    return 0;
}