#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define K 3
#define BLOCK_SIZE 256

__constant__ float d_kernel[K];

__global__ void convConst(float *d_in, float *d_out, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    float sum = 0.0f;

    for (int i = 0; i < K; i++) {
        int idx = tid - (K/2) + i;
        if (idx >= 0 && idx < n) {
            sum += d_in[idx] * d_kernel[i];
        }
    }

    d_out[tid] = sum;
}

int main() {
    int n;
    printf("Enter the size of the input array: ");
    scanf("%d", &n);

    float *h_in = (float*)malloc(n * sizeof(float));
    float *h_out = (float*)malloc(n * sizeof(float));
    float *d_in, *d_out;
    hipMalloc((void**)&d_in, n * sizeof(float));
    hipMalloc((void**)&d_out, n * sizeof(float));

    printf("Enter the input array:\n");
    for (int i = 0; i < n; i++) {
        scanf("%f", &h_in[i]);
    }

    float h_kernel[K] = {1.0f, 2.0f, 3.0f};
    hipMemcpyToSymbol(HIP_SYMBOL(d_kernel), h_kernel, K * sizeof(float));

    int num_blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    hipMemcpy(d_in, h_in, n * sizeof(float), hipMemcpyHostToDevice);
    convConst<<<num_blocks, BLOCK_SIZE>>>(d_in, d_out, n);
    hipMemcpy(h_out, d_out, n * sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken: %f ms\n", milliseconds);

    printf("Output array:\n");
    for (int i = 0; i < n; i++) {
        printf("%f ", h_out[i]);
    }
    printf("\n");

    free(h_in);
    free(h_out);
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}
